#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
// #include <sys/time.h>
#include <sys/timeb.h>
#include <GLFW/glfw3.h>

#define N 1000
#define MIN(x,y) (x<y ? x:y)
#define SQR(x) (x*x)
#define START_END(func, name) (printf("Inicio de %s\n", name), func, printf("Fim de %s\n", name))

#define WIDTH 260
#define HEIGHT WIDTH
#define N_PARTICULAS 128

#define REDCT_NUMTHREADS 256

#define NUMBLOCKS 32

typedef struct Particula {
  double x, y, vx, vy;
} t_particula;

// Tipo que armazena configurações do plot da fractal na janela
typedef struct window_plot {
    double zoom;
    double XOffset;
    double YOffset;
    int width, height;
} t_plot;

// GLubyte PixelBuffer[WIDTH * HEIGHT * 3];

// Atribuindo configurações do programa
t_plot WindowMatrixPlot = {1, -0, 0, WIDTH, HEIGHT};

// Atribui ao pixel na posição (x,y) a cor [r,g,b]
void makePixel(int x, int y, int r, int g, int b, GLubyte* pixels, int width, int height) {
  if (0 <= x && x < width && 0 <= y && y < height) {
    int position = (x + y * width) * 3;
    pixels[position] = r;
    pixels[position + 1] = g;
    pixels[position + 2] = b;
  }
}


//Spacial Data
double xlen; //
double ylen;
int imax;
int jmax;
double delx;
double dely;

//Time Data
double ttime = 0;
double final_time;
double del_time;
double tau; //factor for time step control

//Pressure Data
int max_iter; //max numer of presssure iterations for a time step
int iter; // SOR iter counter
double res; //norm of pressure equation residual
double eps; //stopping tolerance eps for pressure iteration
double omg; //relaxation parameter u> for SOR iteration
double gam; //upwind differencing factor 


//Problem dependent Data
double Re;
double gx;
double gy;
double vx_init;
double vy_init;
double p_init;
int wW,wE,wN,wS; /*specify the type of boundary condition along the
					western (left), eastern (right), northern (upper), and
					southern (lower) boundaries of 17 = [0,xlength] x
					[0,ylength]; each may have one of the values:
					1 for free-slip conditions,
					2 for no-slip conditions,*/
char problem[N];

// struct timeval start, end;
struct timeb start, end, aux_start;
int state = 0;
int n_iter = 5;

// Recebe um valor pertencente a um intervalo [min,max] e retorna o valor transformado
// para o intervalo [floor,ceil]
double map(double value, double min, double max, double floor, double ceil) {
    return floor + (ceil - floor) * ((value - min) / (max - min));
}

// void render(double time, t_particula *particulas) {
//   int tam_p = 0;
// 	double bright = map(time, 0, final_time, 16, 255);

//   for(int z = 0; z < N_PARTICULAS; z++) {
//     t_particula p = particulas[z];
//     int x = p.x;
//     int y = HEIGHT - p.y;

//     for(int i = -tam_p; i <= tam_p; i++) {
//       for(int j = -tam_p; j <= tam_p; j++) {
//         makePixel(
//           x+i, y+j, 
//           bright, bright, bright,
//           PixelBuffer, WIDTH, HEIGHT
//         );
//       }
//     }
//   }
// }

// void atualiza_particulas(double dt, int n_dim, t_particula* particulas, double* d_vx, double* d_vy) {
//   for(int i = 0; i < N_PARTICULAS; i++) {
//     t_particula p = particulas[i];

//     int x = p.x;
//     int y = HEIGHT - p.y;

//     // Limpa o espaco em que a particula estava
//     for(int k = -1; k <= 1; k++) {
//       for(int j = -1; j <= 1; j++) {
//         makePixel(
//           x+k, y+j, 
//           0, 0, 0,
//           PixelBuffer, WIDTH, HEIGHT
//         );
//       }
//     }

//     //POINT pvel = velocidades[WIDTH-(int)p.y][(int)p.x];
// 		int idx = (WIDTH-4)*((int)p.y) + (int)p.x;
//     particulas[i].vx = d_vx[idx];
//     particulas[i].vy = d_vy[idx];

//     particulas[i].x += d_vx[idx] * dt;
//     if(particulas[i].x < 1) particulas[i].x = 1;
//     if(particulas[i].x > n_dim-1) particulas[i].x = n_dim-1;

//     particulas[i].y += d_vy[idx] * dt;
//     if(particulas[i].y < 1) particulas[i].y = 1;
//     if(particulas[i].y > n_dim-1) particulas[i].y = n_dim-1;
//   }
// }

double absf(double a){
	return a < 0 ? -1*a : a;
}

void print_values(){
	printf("Printing values:\n");
	printf("xlen=%f\n",xlen);
	printf("ylen=%f\n",ylen);
	printf("imax=%d\n",imax);
	printf("jmax=%d\n",jmax);
	printf("delx=%f\n",delx);
	printf("dely=%f\n",dely);
	printf("final_time=%f\n",final_time);
	printf("del_time=%f\n",del_time);
	printf("tau=%f\n",tau);
	printf("max_iter=%d\n",max_iter);
	printf("res=%f\n",res);
	printf("eps=%f\n",eps);
	printf("omg=%f\n",omg);
	printf("gam=%f\n",gam);
	printf("Re=%f\n",Re);
	printf("gx=%f\n",gx);
	printf("gy=%f\n",gy);
	printf("vx_init=%f\n",vx_init);
	printf("vy_init=%f\n",vy_init);
	printf("p_init=%f\n",p_init);
	printf("wW=%d\n",wW);
	printf("wE=%d\n",wE);
	printf("wN=%d\n",wW);
	printf("wS=%d\n",wS);
	printf("problem=%s\n",problem);
	printf("-------------------------------\n\n");
}

void read_file(char * file_name){
	FILE *fp;
	char ch[50];
	char* s,*e;
	int num_param = 0;
	fp = fopen(file_name, "r"); // read mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	fscanf(fp,"%s", ch);
	while(!feof(fp)){ 
		s = strtok(ch,":");
		switch(num_param){
			case 0:
				xlen = strtod(s,&e);
				break;
			case 1:
				ylen = strtod(s,&e);
				break;
			case 2:
				imax = strtod(s,&e);
				break;
			case 3:
				jmax = strtod(s,&e);
				break;
			case 4:
				delx = strtod(s,&e);
				if(delx == 0){
					delx = xlen/imax;
				}
				break;
			case 5:
				dely = strtod(s,&e);
				if(dely == 0){
					dely = ylen/jmax;
				}
				break;
			case 6:
				final_time = strtod(s,&e);
				break;
			case 7:
				del_time = strtod(s,&e);
				break;
			case 8:
				tau = strtod(s,&e);
				break;
			case 9:
				max_iter = strtod(s,&e);
				break;
			case 10:
				res = strtod(s,&e);
				break;
			case 11:
				eps = strtod(s,&e);
				break;
			case 12:
				omg = strtod(s,&e);
				break;
			case 13:
				gam = strtod(s,&e);
				break;
			case 14:
				Re = strtod(s,&e);
				break;
			case 15:
				gx = strtod(s,&e);
				break;
			case 16:
				gy = strtod(s,&e);
				break;
			case 17:
				vx_init = strtod(s,&e);
				break;
			case 18:
				vy_init = strtod(s,&e);
				break;
			case 19:
				p_init = strtod(s,&e);
				break;
			case 20:
				wW = strtod(s,&e);
				break;
			case 21:
				wE = strtod(s,&e);
				break;
			case 22:
				wN = strtod(s,&e);
				break;
			case 23:
				wS = strtod(s,&e);
				break;
			case 24:
				strcpy(problem,s);
				break;	
			default:			
				exit(666);
				break;
			}
		num_param++;
		fscanf(fp,"%s", ch);
	}
	fclose(fp);
}

double *vx;
double *vy;
double *p;
double *rhs;
double *F;
double *G;

double *d_vx = NULL;
double *d_vy = NULL;
double *d_p = NULL;
double *d_p_prev = NULL;
double *d_partial = NULL;
double *d_p_diff = NULL;

double *d_rhs = NULL;
double *d_F = NULL;
double *d_G = NULL;
double *d_flag = NULL;

double *d_diag_n = NULL;
double *d_diag_s = NULL;
double *d_diag_w = NULL;
double *d_diag_e = NULL;
double *d_diag_p = NULL; 

double *d_maxdiff = NULL;

double *d_vxdiff = NULL;
double *d_vydiff = NULL;
double *d_vxflag = NULL;
double *d_vyflag = NULL;

int *d_res = NULL;

double *dp=NULL;
	

int n_threads,n_blocos ;

hipError_t err = hipSuccess;

void write_file(char* output){
	FILE *fp;
	int i, j, idx;
	fp = fopen(output, "w"); // write mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	
	// fprintf(fp,"Time taken: %ld seconds\n", stop-start);
	// fprintf(fp,"Simulation Time: %.5f seconds\n", ttime);

	for(i = 0; i < imax+2; i++){
		for(j = 0; j < jmax+2; j++){
			// int ni = imax+1 - i;
			// int nj = jmax+1 - j;
			idx = i*(imax+2)+j;
			fprintf(fp,"vx[%d][%d]=%.10f\n",i,j,vx[idx]);
			fprintf(fp,"vy[%d][%d]=%.10f\n",i,j,vy[idx]);
			// fprintf(fp,"F[%d][%d]=%.10f\n",i,j,F[idx]);
			// fprintf(fp,"G[%d][%d]=%.10f\n",i,j,G[idx]);
			// fprintf(fp,"p[%d][%d]=%.10f\n",i,j,p[idx]);
		}
		if(i < imax+1) fprintf(fp,"\n");
	}
	fclose(fp);
}

void alocate_vectors_host(){
	vx = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	vy = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	p = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	rhs = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	F = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	G = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	
	if(vx == NULL || vy == NULL || p == NULL || rhs == NULL || F == NULL || G == NULL){
		printf("It wasn't possible to alocate memory\n");
		exit(0);
	}
}

void alocate_vectors_device(){	
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMalloc((void **)&d_vx, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector vx (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_vy, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector vy (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_p, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
  err = hipMalloc((void **)&d_p_prev, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device previous vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_p_diff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device diffential vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_partial, NUMBLOCKS * sizeof(double));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device partial vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_rhs, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector rhs (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_F, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector F (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_G, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector G (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
  err = hipMalloc((void **)&d_diag_n, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_n (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	err = hipMalloc((void **)&d_diag_s, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_s (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_w, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_w (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_e, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_e (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_p, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_flag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_flag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vxflag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_vxflag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vyflag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_vyflag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	err = hipMalloc((void **)&d_res, sizeof(int));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer res (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_maxdiff, sizeof(double));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vxdiff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vydiff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void free_vectors_host(){
	free(vx);
	free(vy);
	free(p);
	free(rhs);
	free(F);
	free(G);	
}

void free_vectors_device(){
	
	err = hipFree(d_vx);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector vx (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vy);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector vy (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_p);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_p_prev);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device previous vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_rhs);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector rhs (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_F);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector F (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_G);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector G (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_res);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer res (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	
	err = hipFree(d_maxdiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vxdiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vydiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}	
}

void copy_vectors_host_to_device(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vx from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vy from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector p from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_rhs, rhs, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector rhs from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector F from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector G from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}   
}

void copy_vectors_device_to_host(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vx from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vy from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector p from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(rhs, d_rhs, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector rhs from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(F, d_F, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector F from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector G from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

__device__ double d_absf(double x){
	return x < 0 ? -1*x : x;
}

__global__ void init_UVP(int imax, int jmax, double vx_init, double vy_init, double p_init, double* d_vx, double* d_vy, double* d_p, double* d_rhs, double* d_F, double* d_G){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n){
		d_vx[idx] = vx_init;
		d_vy[idx] = vy_init;
		d_p[idx] = p_init;
		d_rhs[idx] = 0;
		d_F[idx] = 0;
		d_G[idx] = 0;
	}
}

__global__ void dt_reductionMax(int imax, int jmax, double* d_partial, double* d_v){
	__shared__ 	double cache [REDCT_NUMTHREADS];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i, k;
	for(i = idx; i < n; i += inc){
		if(i > (imax+1) && i < (jmax+1)*(imax+2) && i%(imax+2) > 0 && i%(imax+2) < (imax+1)){
			if(temp < d_absf(d_v[i])){
				temp = d_absf(d_v[i]);
			}
		}
	}
	cache[threadIdx.x] = temp;
	__syncthreads();
	
	for(k = (blockDim.x >> 1); k > 0; k >>= 1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0){
		d_partial[blockIdx.x] = cache[0];
	}
}

void comp_delt(){
	int i;
	double aux;
	double aux2;
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	double maxvx = 0;
	double maxvy = 0;
	if(tau > 0){		
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax, jmax, d_partial, d_vx);

		err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vx (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax, jmax, d_partial, d_vy);
		
		err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vy (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		for(i = 0; i < NUMBLOCKS; i++){	
			if(partialvx[i] > maxvx) maxvx = partialvx[i];
			if(partialvy[i] > maxvy) maxvy = partialvy[i];
		}

		aux = MIN((delx/maxvx), (dely/maxvy));
		aux2 = MIN(aux, ((Re/2)*(SQR(delx)*SQR(dely)/(SQR(delx)+SQR(dely)))));
		del_time = tau*aux2;
	}
}

__global__ void set_NorthBond(int imax, int jmax, int wN, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wN){
		case 1://free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] = d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}
			break;			
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] =- d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}	
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] = d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}
			break;
	}
}

__global__ void set_SouthBond(int imax, int jmax, int wS, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wS){
		case 1: //free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] = d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx] = 0;
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] =- d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax+1)*(imax+2)+idx] = 0;
			}
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] = d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx] = 0;
			}
			break;	
	}
}

__global__ void set_WestBond(int imax, int jmax, int wW, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wW){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] = d_vy[idx];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] =- d_vy[idx];
			}			
			break;
		default:
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] = d_vy[idx];
			}
			break;
	}
}

__global__ void set_EastBond(int imax, int jmax, int wE, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wE){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] = d_vy[idx-1];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] =- d_vy[idx-1];
			}
			break;
		default:
				if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] = d_vy[idx-1];
			}
			break;
	}
}

void set_bondCond(){
	set_NorthBond<<< n_blocos, n_threads >>>(imax, jmax, wN, d_vx, d_vy);
	set_SouthBond<<< n_blocos, n_threads >>>(imax, jmax, wS, d_vx, d_vy);
	set_WestBond<<< n_blocos, n_threads >>>(imax, jmax, wW, d_vx, d_vy);
	set_EastBond<<< n_blocos, n_threads >>>(imax, jmax, wE, d_vx, d_vy);
}

__global__ void set_lidDrivenCavityProblem(double lid_vel, int imax, int jmax, double* d_vx){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n && idx > 0 && idx < imax+1){
		d_vx[idx] = 2.0*lid_vel - d_vx[imax+2+idx];
	}
}

__device__ double del_vx_sqr_del_x(double gam, double delx, double vx_C_point, double vx_W_point, double vx_E_point){ //d(vx²)/dx
	double aws;
	aws = (SQR((vx_C_point+vx_E_point)/2) - SQR((vx_W_point+vx_C_point)/2))/delx;
	aws += (gam/delx)*((d_absf(vx_C_point+vx_E_point)/2)*((vx_C_point-vx_E_point)/2) - (d_absf(vx_W_point+vx_C_point)/2)*((vx_W_point-vx_C_point)/2));
	return aws;
}

__device__ double del_vx_vy_del_y(double gam, double dely, double vx_N_point, double vx_C_point, double vx_S_point, double vy_C_point, double vy_E_point, double vy_S_point, double vy_SE_point){ //d(vx*vy)/dy
	double aws;
	aws = ((vy_C_point+vy_E_point)*(vx_C_point+vx_N_point) - (vy_S_point+vy_SE_point)*(vx_S_point+vx_C_point))/2*dely;
	aws += (gam/2*dely)*(d_absf(vy_C_point+vy_E_point)*(vx_C_point-vx_N_point) - d_absf(vy_S_point+vy_SE_point)*(vx_S_point-vx_C_point));
	return aws;
}

__device__ double del_sqr_vx_del_sqr_x(double delx, double vx_E_point, double vx_C_point, double vx_W_point){ //d²(vx)/dx²
	return (vx_E_point-2*vx_C_point+vx_W_point)/SQR(delx);
}

__device__ double del_sqr_vx_del_sqr_y(double dely, double vx_N_point, double vx_C_point, double vx_S_point){ //d²(vx)/dy²
	return (vx_N_point-2*vx_C_point+vx_S_point)/SQR(dely);
}

__device__ double del_vy_sqr_del_y(double gam, double dely, double vy_N_point, double vy_C_point, double vy_S_point){ // d(vy²)/dy
	double aws;
	aws = (SQR((vy_C_point+vy_N_point)/2) - SQR((vy_S_point+vy_C_point)/2))/dely;
	aws += (gam/dely)*(d_absf((vy_C_point+vy_N_point)/2)*((vy_C_point-vy_N_point)/2) - d_absf((vy_S_point+vy_C_point)/2)*((vy_S_point-vy_C_point)/2));
	return aws;
}

__device__ double del_vy_vx_del_x(double gam, double delx, double vy_W_point, double vy_C_point, double vy_E_point, double vx_N_point, double vx_C_point, double vx_W_point, double vx_NW_point){ //d(vx*vy)/dx
	double aws;
	aws = ((vx_C_point+vx_N_point)*(vy_C_point+vy_E_point) - (vx_W_point+vx_NW_point)*(vy_W_point+vy_C_point))/2*delx;
	aws += (gam/2*delx)*(d_absf(vx_C_point+vx_N_point)*(vy_C_point-vy_E_point) - d_absf(vx_W_point+vx_NW_point)*(vy_W_point-vy_C_point));
	return aws;
}

__device__ double del_sqr_vy_del_sqr_x(double delx, double vy_E_point, double vy_C_point, double vy_W_point){//d²(vy)/dx²
	return (vy_E_point-2*vy_C_point+vy_W_point)/SQR(delx);
}

__device__ double del_sqr_vy_del_sqr_y(double dely, double vy_N_point, double vy_C_point, double vy_S_point){//d²(vy)/dy²
	return (vy_N_point-2*vy_C_point+vy_S_point)/SQR(dely);
}

__global__ void comp_FG(int imax, int jmax, double gam, double delx, double dely, double Re, double gx, double gy, double del_time, double* d_vx, double* d_vy, double* d_F, double* d_G){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		double term1 = del_sqr_vx_del_sqr_x(delx, d_vx[idx+1], d_vx[idx], d_vx[idx-1]); 
								// del_sqr_vx_del_sqr_x(matrix[j][i+1].vx, matrix[j][i].vx,matrix[j][i-1].vx);
				
		double term2 = del_sqr_vx_del_sqr_y(dely, d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx+(imax+2)]); 
								// del_sqr_vx_del_sqr_y(matrix[j-1][i].vx, matrix[j][i].vx,matrix[j+1][i].vx);
				
		double term3 = del_vx_sqr_del_x(gam, delx, d_vx[idx], d_vx[idx-1], d_vx[idx+1]);
								// del_vx_sqr_del_x(matrix[j][i].vx,matrix[j][i-1].vx,matrix[j][i+1].vx);
				
		double term4 = del_vx_vy_del_y(gam, dely, d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx+(imax+2)], d_vy[idx], d_vy[idx+1], d_vy[idx+(imax+2)], d_vy[idx+(imax+2)+1]); 
								// del_vx_vy_del_y(matrix[j-1][i].vx,matrix[j][i].vx,matrix[j+1][i].vx, matrix[j][i].vy,matrix[j][i+1].vy,matrix[j+1][i].vy,matrix[j+1][i+1].vy);
		
		d_F[idx] = d_vx[idx] + del_time*(((term1+term2)/Re) - term3 - term4 + gx);
		//matrix[j][i].F=matrix[j][i].vx + del_time*(((term1+term2)/Re) - term3 - term4 + gx);	
	
	} else if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && (idx%(imax+2) == 0 || idx%(imax+2) == (imax+1)) ) {
		d_F[idx] = d_vx[idx]; //matrix[j][imax].F=matrix[j][imax].vx;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		
		double term5 = del_sqr_vy_del_sqr_x(delx, d_vy[idx+1], d_vy[idx], d_vy[idx-1]);		
								// del_sqr_vy_del_sqr_x(matrix[j][i+1].vy,matrix[j][i].vy,matrix[j][i-1].vy);
		
		double term6 = del_sqr_vy_del_sqr_y(dely, d_vy[idx-(imax+2)], d_vy[idx], d_vy[idx+(imax+2)]); 
								// del_sqr_vy_del_sqr_y(matrix[j-1][i].vy,matrix[j][i].vy,matrix[j+1][i].vy);
		
		double term7 = del_vy_vx_del_x(gam, delx, d_vy[idx-1], d_vy[idx], d_vy[idx+1], d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx-1], d_vx[idx-(imax+2)-1]); 
								// del_vy_vx_del_x(matrix[j][i-1].vy, matrix[j][i].vy,matrix[j][i+1].vy, matrix[j-1][i].vx,matrix[j][i].vx, matrix[j][i-1].vx ,matrix[j-1][i-1].vx);
				
		double term8 = del_vy_sqr_del_y(gam, dely, d_vy[idx-(imax+2)], d_vy[idx], d_vy[idx+(imax+2)]);
								// del_vy_sqr_del_y(matrix[j-1][i].vy ,matrix[j][i].vy,matrix[j+1][i].vy);
		
		d_G[idx] = d_vy[idx] + del_time*(((term5+term6)/Re) - term7 - term8 + gy);
	
	} else if(idx < n && idx > (imax+1) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_G[idx] = d_vy[idx];
	}	
}

__global__ void comp_RHS(int imax, int jmax, double delx, double dely, double del_time, double* d_rhs, double* d_F, double* d_G){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_rhs[idx] = ((d_F[idx]-d_F[idx-1])/delx +(d_G[idx]-d_G[idx+(imax+2)])/dely)/del_time;
	}	
}

__global__ void build_poisson_system(int jmax, int imax, double delx, double dely, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p){
	int i, j, idx;
	int ew, ee, es, en;
	for(j = jmax; j > 0; j--){
		es = j<jmax ? 1 : 0;
		en = j>1 ? 1 : 0;
		for(i = 1; i < imax+1; i++){
			idx = j*(imax+2)+i;
			ew = i>1 ? 1 : 0;
			ee = i<imax ? 1 : 0;
			d_diag_e[idx] = ee/SQR(delx);
			d_diag_w[idx] = ew/SQR(delx);
			d_diag_p[idx] = ((ee+ew)/SQR(delx) + (en+es)/SQR(dely));
			d_diag_n[idx] = en/SQR(dely);
			d_diag_s[idx] = es/SQR(dely);			
		}
	}
}

__global__ void reductionMax(int imax, int jmax, double* d_partial, double* d_diff, double* d_flag){
	__shared__ 	double cache [REDCT_NUMTHREADS];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i, k;
	
	for(i = idx; i < n; i += inc){
		if(d_flag[i]){
			if(temp < d_diff[i]) temp = d_diff[i];
		}
	}
	
	cache[threadIdx.x] = temp;
	__syncthreads();
	
	for(k = (blockDim.x >> 1); k > 0; k >>= 1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0){
		d_partial[blockIdx.x] = cache[0];
	}
}

__global__ void red_SOR(int imax, int jmax, double omg, double* d_p, double* d_rhs, double* d_p_diff, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p, double* d_p_prev, double* d_flag){	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int size = (imax+2)*(jmax+2);
	int line = idx / (imax+2);
	int paridade = line % 2;
	double aux;
	
	if(	idx < size && idx > (imax+1) && idx < (imax+2)*(jmax+1) && 
			idx % 2 == paridade && idx % (imax+2) > 0 && idx % (imax+2) < (imax+1)){
		d_p_prev[idx] = d_p[idx];
		aux = d_diag_s[idx]*d_p[idx+(imax+2)] + d_diag_n[idx]*d_p[idx-(imax+2)]+ d_diag_e[idx]*d_p[idx+1] + d_diag_w[idx]*d_p[idx-1];
		aux = (1-omg)*d_p[idx] + omg*(aux-d_rhs[idx])/d_diag_p[idx];
		d_p_diff[idx] = d_absf(aux-d_p[idx]);		
		d_flag[idx] = 1;
		d_p[idx] = aux;
	}	else{
		d_flag[idx] = 0;
	}
}

__global__ void black_SOR(int imax, int jmax, double omg, double* d_p, double* d_rhs, double* d_p_diff, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p, double* d_p_prev, double* d_flag){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int size = (imax+2)*(jmax+2);
	int line = idx / (imax+2);
	int paridade =!(line % 2);
	double aux;
		
	if(	idx < size && idx > (imax+1) && idx < (imax+2)*(jmax+1) && 
			idx % 2 == paridade && idx % (imax+2) > 0 && idx % (imax+2) < (imax+1)){
		d_p_prev[idx] = d_p[idx];
		aux = d_diag_s[idx]*d_p[idx+(imax+2)] + d_diag_n[idx]*d_p[idx-(imax+2)]+d_diag_e[idx]*d_p[idx+1]+d_diag_w[idx]*d_p[idx-1];
		aux = (1-omg)*d_p[idx] + omg*(aux-d_rhs[idx])/d_diag_p[idx];
		d_p_diff[idx] = d_absf(aux-d_p[idx]);		
		d_flag[idx] = 1;
		d_p[idx] = aux;
	}	else{
		d_flag[idx] = 0;
	}	
}

int Poisson(){
	int iter = 0;
	int i;
	double diff = 0;
	double partial[NUMBLOCKS];
	struct timeb start_rbsor, end_rbsor, start_redct, end_redct, start_aux, end_aux, start_total, end_total;
	int total_rbsor = 0, total_redct = 0, total_aux = 0, total = 0;

	ftime(&start_total);
	while(iter < max_iter){
		ftime(&start_rbsor);
		for(i = 0; i < n_iter; i++){
			red_SOR<<<n_blocos, n_threads>>>(imax, jmax, omg, d_p, d_rhs, d_p_diff, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p, d_p_prev, d_flag);
			black_SOR<<<n_blocos, n_threads>>>(imax, jmax, omg, d_p, d_rhs, d_p_diff, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p, d_p_prev, d_flag);
		}
		ftime(&end_rbsor);
		total_rbsor += (int) (1000.0 * (end_rbsor.time - start_rbsor.time)
        + (end_rbsor.millitm - start_rbsor.millitm));
		
		diff = 0;

		ftime(&start_redct);
		reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_p_diff, d_flag);
		ftime(&end_redct);
		total_redct += (int) (1000.0 * (end_redct.time - start_redct.time)
        + (end_redct.millitm - start_redct.millitm));
		

		// if((iter/n_iter) % 1000 != 0) {
		// 	iter += n_iter;
		// 	continue;
		// }
		ftime(&start_aux);
		err = hipMemcpy(partial, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		total_aux += (int) (1000.0 * (end_aux.time - start_aux.time)
        + (end_aux.millitm - start_aux.millitm));
		
		for(i = 0; i < NUMBLOCKS; i++){
			if(partial[i] > diff) diff = partial[i];
		}
		ftime(&end_aux);
		iter += n_iter;
		
		if(diff < eps){
			return iter;
		}
	}
	ftime(&end_total);
	total += (int) (1000.0 * (end_total.time - start_total.time)
				+ (end_total.millitm - start_total.millitm));

	printf("Total time - reductionMax: %d milliseconds\n", total_redct);
	printf("Total time - redBlackSOR: %d milliseconds\n", total_rbsor);
	printf("Total time - aux: %d milliseconds\n", total_aux);
	printf("Total time - total: %d milliseconds\n", total);
	return iter;
}

__global__ void d_adap_Vel(int imax, int jmax, double delx, double dely, double del_time, double* d_vx, double* d_vy, double* d_p, double* d_F, double* d_G, double* d_vxdiff, double* d_vydiff, double* d_vxflag, double* d_vyflag){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int n = (imax+2)*(jmax+2);
	
	double aux;
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		aux = d_F[idx] - (del_time*(d_p[idx+1] - d_p[idx])/delx); //matrix[j][i].F - (del_time*(matrix[j][i+1].p - matrix[j][i].p)/delx);
		d_vxdiff[idx] = d_absf(aux-d_vx[idx]);   //absf(aux-matrix[j][i].vx);
		d_vxflag[idx] = 1;
		d_vx[idx] = aux;		
	} else{
		d_vxflag[idx] = 0;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		aux = d_G[idx] - (del_time*(d_p[idx-(imax+2)] - d_p[idx])/dely); //matrix[j][i].G - (del_time*(matrix[j-1][i].p - matrix[j][i].p)/dely);
		d_vydiff[idx] = d_absf(aux-d_vy[idx]);//absf(aux-matrix[j][i].vy);
		d_vyflag[idx] = 1;
		d_vy[idx] = aux;
	}	else{
		d_vyflag[idx] = 0;
	}	
}

int adap_Vel(int n_blocos, int n_threads){
	double diffvx = 0;
	double diffvy = 0;
	int i;
	
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	
	d_adap_Vel<<< n_blocos, n_threads >>>(imax, jmax, delx, dely, del_time, d_vx, d_vy, d_p, d_F, d_G, d_vxdiff, d_vydiff, d_vxflag, d_vyflag);		

	reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_vxdiff, d_vxflag);
	err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_vydiff, d_vyflag);
	err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	for(i = 0; i < NUMBLOCKS; i++){
		if(partialvy[i] > diffvy) diffvy = partialvy[i];
		if(partialvx[i] > diffvx) diffvx = partialvx[i];
	}
	
	if(diffvy < eps && diffvx < eps) return 1;
	return 0;
}


int main(int argc, char ** argv){
	// hipEventCreate(&start);
	// hipEventCreate(&stop);

	read_file(argv[1]);
	alocate_vectors_host();
	alocate_vectors_device();
	copy_vectors_host_to_device();
    
	n_threads = (imax+2);
	n_blocos = ((imax+2)*(jmax+2)+ n_threads-1)/n_threads;
	init_UVP<<<n_blocos, n_threads>>>(imax, jmax, vx_init, vy_init, p_init, d_vx, d_vy, d_p, d_rhs, d_F, d_G);
	build_poisson_system<<<1, 1>>>(jmax, imax, delx, dely, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p);
	
	int set_time = 1;
	double ant_del_time = 1.0;
	double eps_time = 1e-7;
	int num_time = 0;
	int limit = 100;
	int frames = 0;
	int max_frames = 2000;

  // gettimeofday(&start, NULL);
	ftime(&start);

	t_particula particulas[N_PARTICULAS];
  for(int i = 0; i < N_PARTICULAS; i++) {
    int sq = (int)sqrt(N_PARTICULAS);

    particulas[i].x = 2+(i % sq) * (WIDTH-2)/sq;
    particulas[i].y = 2+(i / sq) * (HEIGHT-2)/sq;
    particulas[i].vx = 0;
    particulas[i].vy = 0;
  }

/*
  GLFWwindow* window;
  // Inicializando a biblioteca
  if (!glfwInit())
    return -1;

  // Criando a janela e seu contexto OpenGL
  window = glfwCreateWindow(WIDTH, HEIGHT, "Visualizacao de movimento de particulas dada equações de Navier-Stokes", NULL, NULL);
  if (!window){
    glfwTerminate();
    return -1;
  }

  // Cria o contexto atual da janela
  glfwMakeContextCurrent(window);

*/
	int frame = 0;
	float last_time_frame = 0.0;
	printf("Entrando no laco principal\n");
	while(!state){
	/*
		// Configuração da visualização
    glfwGetFramebufferSize(window, &WindowMatrixPlot.width, &WindowMatrixPlot.height);
    glViewport(0, 0, WindowMatrixPlot.width, WindowMatrixPlot.height);

    // Atualiza a posicao de cada uma das particulas
		copy_vectors_device_to_host();
    atualiza_particulas(ant_del_time, WIDTH, particulas, vx, vy);

    // Pinta os pixels na janela
    render(ttime, particulas);

		frame++;
		glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, PixelBuffer);

    // Funções necesárias para o funcionamento da biblioteca que desenha os pixels
    glfwSwapBuffers(window);
    glfwPollEvents();
	*/

		if(set_time){
			comp_delt();
		}
		if(absf(ant_del_time-del_time) < eps_time){
			num_time++;
			if(num_time == limit){
				set_time = 0;
			}
		}	else{
			num_time = 0;
		}
		
		set_bondCond();
		set_lidDrivenCavityProblem<<<n_blocos, n_threads>>>(1.0, imax, jmax, d_vx);
		
		comp_FG<<<n_blocos, n_threads>>>(imax, jmax, gam, delx, dely, Re, gx, gy, del_time, d_vx, d_vy, d_F, d_G);
		comp_RHS<<<n_blocos, n_threads>>>(imax, jmax, delx, dely, del_time, d_rhs, d_F, d_G);		

		ftime(&aux_start);
		printf("%d iteracoes \t", Poisson());
		ftime(&end);
		int time_frame = (int) (1000.0 * (end.time - aux_start.time)
        + (end.millitm - aux_start.millitm));
		printf("Time elapsed - Poisson: %lf seconds, frame %d\n\n", time_frame/1000.0, frames);
		if(time_frame < 1000) break;
		// printf("Time = %lf/%lf\r", ttime, final_time);
		state = adap_Vel(n_blocos, n_threads);
		
		ttime += del_time;
		ant_del_time = del_time;

		frame++; frames++;
		ftime(&end);
		time_frame = (int) (1000.0 * (end.time - start.time)
        + (end.millitm - start.millitm));

		// printf("Time = %d\n", time_frame);
		if(time_frame - last_time_frame >= 1000.0) {
			// printf("%d fps\n", frame);
			frame = 0;
			last_time_frame = time_frame;
		}
	}

	set_bondCond();		
	set_lidDrivenCavityProblem<<<n_blocos, n_threads>>>(1.0, imax, jmax, d_vx);
	// gettimeofday(&end, NULL);
	ftime(&end);

	copy_vectors_device_to_host(); 
  write_file(argv[2]);
    
  free_vectors_device();	
	free_vectors_host();

	int milliseconds = (int) (1000.0 * (end.time - start.time) + (end.millitm - start.millitm));
	
	printf("Time = %lf/%lf\n", ttime, final_time);
	printf("Time elapsed: %lf seconds\n", milliseconds/1000.0);

	/*
	frame = 0;
	while(frame < max_frames){
		// Configuração da visualização
    glfwGetFramebufferSize(window, &WindowMatrixPlot.width, &WindowMatrixPlot.height);
    glViewport(0, 0, WindowMatrixPlot.width, WindowMatrixPlot.height);

    // Atualiza a posicao de cada uma das particulas
    atualiza_particulas(ant_del_time, WIDTH, particulas, vx, vy);

    // Pinta os pixels na janela
    render(ttime, particulas);

		frame++;
		glDrawPixels(WIDTH, HEIGHT, GL_RGB, GL_UNSIGNED_BYTE, PixelBuffer);

    // Funções necesárias para o funcionamento da biblioteca que desenha os pixels
    glfwSwapBuffers(window);
    glfwPollEvents();
	}
	*/
	return 0;
}
