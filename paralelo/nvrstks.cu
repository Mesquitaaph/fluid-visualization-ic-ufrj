#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
// #include <sys/time.h>
#include <sys/timeb.h>
#include <GLFW/glfw3.h>

#define N 1000
#define MIN(x,y) (x<y ? x:y)
#define SQR(x) (x*x)
#define START_END(func, name) (printf("Inicio de %s\n", name), func, printf("Fim de %s\n", name))

#define WIDTH 260
#define HEIGHT WIDTH
#define N_PARTICULAS 128

#define REDCT_NUMTHREADS 256

#define NUMBLOCKS 32

#define TAM_BUFFER_SOR 257

typedef struct Particula {
  double x, y, vx, vy;
} t_particula;

// Tipo que armazena configurações do plot da fractal na janela
typedef struct window_plot {
    double zoom;
    double XOffset;
    double YOffset;
    int width, height;
} t_plot;

// GLubyte PixelBuffer[WIDTH * HEIGHT * 3];

// Atribuindo configurações do programa
t_plot WindowMatrixPlot = {1, -0, 0, WIDTH, HEIGHT};

// Atribui ao pixel na posição (x,y) a cor [r,g,b]
void makePixel(int x, int y, int r, int g, int b, GLubyte* pixels, int width, int height) {
  if (0 <= x && x < width && 0 <= y && y < height) {
    int position = (x + y * width) * 3;
    pixels[position] = r;
    pixels[position + 1] = g;
    pixels[position + 2] = b;
  }
}


//Spacial Data
double xlen; //
double ylen;
int imax;
int jmax;
double delx;
double dely;

//Time Data
double ttime = 0;
double final_time;
double del_time;
double tau; //factor for time step control

//Pressure Data
int max_iter; //max numer of presssure iterations for a time step
int iter; // SOR iter counter
double res; //norm of pressure equation residual
double eps; //stopping tolerance eps for pressure iteration
double omg; //relaxation parameter u> for SOR iteration
double gam; //upwind differencing factor 


//Problem dependent Data
double Re;
double gx;
double gy;
double vx_init;
double vy_init;
double p_init;
int wW,wE,wN,wS; /*specify the type of boundary condition along the
					western (left), eastern (right), northern (upper), and
					southern (lower) boundaries of 17 = [0,xlength] x
					[0,ylength]; each may have one of the values:
					1 for free-slip conditions,
					2 for no-slip conditions,*/
char problem[N];

// struct timeval start, end;
struct timeb start, end, aux_start;
int state = 0;
int n_iter = 5;
__device__ int d_n_iter = 5;

// Recebe um valor pertencente a um intervalo [min,max] e retorna o valor transformado
// para o intervalo [floor,ceil]
double map(double value, double min, double max, double floor, double ceil) {
    return floor + (ceil - floor) * ((value - min) / (max - min));
}

double absf(double a){
	return a < 0 ? -1*a : a;
}

void print_values(){
	printf("Printing values:\n");
	printf("xlen=%f\n",xlen);
	printf("ylen=%f\n",ylen);
	printf("imax=%d\n",imax);
	printf("jmax=%d\n",jmax);
	printf("delx=%f\n",delx);
	printf("dely=%f\n",dely);
	printf("final_time=%f\n",final_time);
	printf("del_time=%f\n",del_time);
	printf("tau=%f\n",tau);
	printf("max_iter=%d\n",max_iter);
	printf("res=%f\n",res);
	printf("eps=%f\n",eps);
	printf("omg=%f\n",omg);
	printf("gam=%f\n",gam);
	printf("Re=%f\n",Re);
	printf("gx=%f\n",gx);
	printf("gy=%f\n",gy);
	printf("vx_init=%f\n",vx_init);
	printf("vy_init=%f\n",vy_init);
	printf("p_init=%f\n",p_init);
	printf("wW=%d\n",wW);
	printf("wE=%d\n",wE);
	printf("wN=%d\n",wW);
	printf("wS=%d\n",wS);
	printf("problem=%s\n",problem);
	printf("-------------------------------\n\n");
}

void read_file(char * file_name){
	FILE *fp;
	char ch[50];
	char* s,*e;
	int num_param = 0;
	fp = fopen(file_name, "r"); // read mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	fscanf(fp,"%s", ch);
	while(!feof(fp)){ 
		s = strtok(ch,":");
		switch(num_param){
			case 0:
				xlen = strtod(s,&e);
				break;
			case 1:
				ylen = strtod(s,&e);
				break;
			case 2:
				imax = strtod(s,&e);
				break;
			case 3:
				jmax = strtod(s,&e);
				break;
			case 4:
				delx = strtod(s,&e);
				if(delx == 0){
					delx = xlen/imax;
				}
				break;
			case 5:
				dely = strtod(s,&e);
				if(dely == 0){
					dely = ylen/jmax;
				}
				break;
			case 6:
				final_time = strtod(s,&e);
				break;
			case 7:
				del_time = strtod(s,&e);
				break;
			case 8:
				tau = strtod(s,&e);
				break;
			case 9:
				max_iter = strtod(s,&e);
				break;
			case 10:
				res = strtod(s,&e);
				break;
			case 11:
				eps = strtod(s,&e);
				break;
			case 12:
				omg = strtod(s,&e);
				break;
			case 13:
				gam = strtod(s,&e);
				break;
			case 14:
				Re = strtod(s,&e);
				break;
			case 15:
				gx = strtod(s,&e);
				break;
			case 16:
				gy = strtod(s,&e);
				break;
			case 17:
				vx_init = strtod(s,&e);
				break;
			case 18:
				vy_init = strtod(s,&e);
				break;
			case 19:
				p_init = strtod(s,&e);
				break;
			case 20:
				wW = strtod(s,&e);
				break;
			case 21:
				wE = strtod(s,&e);
				break;
			case 22:
				wN = strtod(s,&e);
				break;
			case 23:
				wS = strtod(s,&e);
				break;
			case 24:
				strcpy(problem,s);
				break;	
			default:			
				exit(666);
				break;
			}
		num_param++;
		fscanf(fp,"%s", ch);
	}
	fclose(fp);
}

double *vx;
double *vy;
double *p;
double *rhs;
double *F;
double *G;

double *d_vx = NULL;
double *d_vy = NULL;
double *d_p = NULL;
double *d_p_red = NULL;
double *d_p_black = NULL;
double *d_p_prev = NULL;
double *d_partial = NULL;
double *d_p_diff = NULL;

double *d_rhs = NULL;
double *d_F = NULL;
double *d_G = NULL;
double *d_flag = NULL;

double *d_diag_n = NULL;
double *d_diag_s = NULL;
double *d_diag_w = NULL;
double *d_diag_e = NULL;
double *d_diag_p = NULL; 

double *d_maxdiff = NULL;

double *d_vxdiff = NULL;
double *d_vydiff = NULL;
double *d_vxflag = NULL;
double *d_vyflag = NULL;

int *d_res = NULL;

double *dp=NULL;


int n_threads,n_blocos ;

hipError_t err = hipSuccess;

texture<double, 1, hipReadModeElementType> tex;

void write_file(char* output){
	FILE *fp;
	int i, j, idx;
	fp = fopen(output, "w"); // write mode

	if (fp == NULL){
	  perror("Error while opening the file.\n");
	  exit(EXIT_FAILURE);
	}
	
	int milliseconds = (int) (1000.0 * (end.time - start.time) + (end.millitm - start.millitm));
	// fprintf(fp,"Time taken: %lf seconds\n", milliseconds/1000.0);
	// fprintf(fp,"Simulation Time: %.5f seconds\n", ttime);

	for(i = 0; i < imax+2; i++){
		for(j = 0; j < jmax+2; j++){
			idx = i*(imax+2)+j;
			fprintf(fp,"vx[%d][%d]=%.10f\n",i,j,vx[idx]);
			fprintf(fp,"vy[%d][%d]=%.10f\n",i,j,vy[idx]);
			// fprintf(fp,"F[%d][%d]=%.10f\n",i,j,F[idx]);
			// fprintf(fp,"G[%d][%d]=%.10f\n",i,j,G[idx]);
			// fprintf(fp,"p[%d][%d]=%.10f\n",i,j,p[idx]);
		}
		if(i < imax+1) fprintf(fp,"\n");
	}
	fclose(fp);
}

void alocate_vectors_host(){
	vx = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	vy = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	p = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	rhs = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	F = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	G = (double*)malloc((imax+2)*(jmax+2) * sizeof(double));
	
	if(vx == NULL || vy == NULL || p == NULL || rhs == NULL || F == NULL || G == NULL){
		printf("It wasn't possible to alocate memory\n");
		exit(0);
	}
}

void alocate_vectors_device(){	
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	size_t red_black_size = (imax+2)*(jmax+2)/2 * sizeof(double);
	
	err = hipMalloc((void **)&d_vx, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector vx (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_vy, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector vy (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_p, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
  err = hipMalloc((void **)&d_p_prev, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device previous vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_p_diff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device diffential vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_partial, NUMBLOCKS * sizeof(double));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device partial vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_rhs, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector rhs (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_F, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector F (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_G, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector G (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
  err = hipMalloc((void **)&d_diag_n, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_n (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    
	err = hipMalloc((void **)&d_diag_s, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_s (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_w, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_w (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_e, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_e (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_diag_p, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector diag_p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_flag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_flag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vxflag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_vxflag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vyflag, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector d_vyflag (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	err = hipMalloc((void **)&d_res, sizeof(int));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer res (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMalloc((void **)&d_maxdiff, sizeof(double));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vxdiff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_vydiff, size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_p_red, red_black_size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer d_p_red (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_p_black, red_black_size);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device pointer d_p_black (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void free_vectors_host(){
	free(vx);
	free(vy);
	free(p);
	free(rhs);
	free(F);
	free(G);
}

void free_vectors_device(){
	
	err = hipFree(d_vx);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector vx (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vy);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector vy (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_p);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_p_prev);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device previous vector p (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_rhs);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector rhs (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_F);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector F (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_G);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device vector G (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_res);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer res (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	
	err = hipFree(d_maxdiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer maxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vxdiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer vxdiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipFree(d_vydiff);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer vydiff (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}	
	
	err = hipFree(d_p_red);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer d_p_red (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}	
	
	err = hipFree(d_p_black);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to free device pointer d_p_black (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}	
}

void copy_vectors_host_to_device(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(d_vx, vx, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vx from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_vy, vy, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vy from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_p, p, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector p from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_rhs, rhs, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector rhs from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_F, F, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector F from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(d_G, G, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector G from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}   
}

void copy_vectors_device_to_host(){
	size_t size = (imax+2)*(jmax+2) * sizeof(double);
	
	err = hipMemcpy(vx, d_vx, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vx from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(vy, d_vy, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector vy from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector p from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(rhs, d_rhs, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector rhs from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(F, d_F, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector F from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	err = hipMemcpy(G, d_G, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy vector G from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

__device__ double d_absf(double x){
	return x < 0 ? -1*x : x;
}

__global__ void init_UVP(int imax, int jmax, double vx_init, double vy_init, double p_init, double* d_vx, double* d_vy, double* d_p, double* d_rhs, double* d_F, double* d_G, double* d_p_red, double* d_p_black){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n){
		d_vx[idx] = vx_init;
		d_vy[idx] = vy_init;
		d_p[idx] = p_init;
		d_rhs[idx] = 0;
		d_F[idx] = 0;
		d_G[idx] = 0;

		if(idx < n/2) {
			d_p_red[idx] = p_init;
			d_p_black[idx] = p_init;
		}
	}
}

__global__ void dt_reductionMax(int imax, int jmax, double* d_partial, double* d_v){
	__shared__ 	double cache [REDCT_NUMTHREADS];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i, k;
	for(i = idx; i < n; i += inc){
		if(i > (imax+1) && i < (jmax+1)*(imax+2) && i%(imax+2) > 0 && i%(imax+2) < (imax+1)){
			if(temp < d_absf(d_v[i])){
				temp = d_absf(d_v[i]);
			}
		}
	}
	cache[threadIdx.x] = temp;
	__syncthreads();
	
	for(k = (blockDim.x >> 1); k > 0; k >>= 1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0){
		d_partial[blockIdx.x] = cache[0];
	}
}

void comp_delt(){
	int i;
	double aux;
	double aux2;
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	double maxvx = 0;
	double maxvy = 0;
	if(tau > 0){		
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax, jmax, d_partial, d_vx);

		err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vx (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		dt_reductionMax<<<REDCT_NUMTHREADS,NUMBLOCKS>>>(imax, jmax, d_partial, d_vy);
		
		err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host vy (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		for(i = 0; i < NUMBLOCKS; i++){	
			if(partialvx[i] > maxvx) maxvx = partialvx[i];
			if(partialvy[i] > maxvy) maxvy = partialvy[i];
		}

		aux = MIN((delx/maxvx), (dely/maxvy));
		aux2 = MIN(aux, ((Re/2)*(SQR(delx)*SQR(dely)/(SQR(delx)+SQR(dely)))));
		del_time = tau*aux2;
	}
}

__global__ void set_NorthBond(int imax, int jmax, int wN, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wN){
		case 1://free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] = d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}
			break;			
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] =- d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}	
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[idx] = d_vx[imax+2+idx];
				d_vy[imax+2+idx] = 0;
			}
			break;
	}
}

__global__ void set_SouthBond(int imax, int jmax, int wS, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wS){
		case 1: //free-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] = d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx] = 0;
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] =- d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax+1)*(imax+2)+idx] = 0;
			}
			break;
		default:
			if(idx < n && idx > 0 && idx < imax+1){
				d_vx[(jmax+1)*(imax+2)+idx] = d_vx[jmax*(imax+2)+idx];
				d_vy[(jmax)*(imax+2)+idx] = 0;
			}
			break;	
	}
}

__global__ void set_WestBond(int imax, int jmax, int wW, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wW){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] = d_vy[idx];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] =- d_vy[idx];
			}			
			break;
		default:
			if(idx < n && idx%(imax+2) == 1 && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx-1] = d_vy[idx];
			}
			break;
	}
}

__global__ void set_EastBond(int imax, int jmax, int wE, double* d_vx, double* d_vy){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	switch(wE){
		case 1: //free-slip condition
			if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] = d_vy[idx-1];
			}
			break;
		case 2://no-slip condition
			if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] =- d_vy[idx-1];
			}
			break;
		default:
				if(idx < n && idx%(imax+2) == (imax+1) && idx > (imax+1) && idx < (n-imax-2)){
				d_vx[idx-1] = 0;
				d_vy[idx] = d_vy[idx-1];
			}
			break;
	}
}

void set_bondCond(){
	set_NorthBond<<< n_blocos, n_threads >>>(imax, jmax, wN, d_vx, d_vy);
	set_SouthBond<<< n_blocos, n_threads >>>(imax, jmax, wS, d_vx, d_vy);
	set_WestBond<<< n_blocos, n_threads >>>(imax, jmax, wW, d_vx, d_vy);
	set_EastBond<<< n_blocos, n_threads >>>(imax, jmax, wE, d_vx, d_vy);
}

__global__ void set_lidDrivenCavityProblem(double lid_vel, int imax, int jmax, double* d_vx){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	if(idx < n && idx > 0 && idx < imax+1){
		d_vx[idx] = 2.0*lid_vel - d_vx[imax+2+idx];
	}
}

__device__ double del_vx_sqr_del_x(double gam, double delx, double vx_C_point, double vx_W_point, double vx_E_point){ //d(vx²)/dx
	double aws;
	aws = (SQR((vx_C_point+vx_E_point)/2) - SQR((vx_W_point+vx_C_point)/2))/delx;
	aws += (gam/delx)*((d_absf(vx_C_point+vx_E_point)/2)*((vx_C_point-vx_E_point)/2) - (d_absf(vx_W_point+vx_C_point)/2)*((vx_W_point-vx_C_point)/2));
	return aws;
}

__device__ double del_vx_vy_del_y(double gam, double dely, double vx_N_point, double vx_C_point, double vx_S_point, double vy_C_point, double vy_E_point, double vy_S_point, double vy_SE_point){ //d(vx*vy)/dy
	double aws;
	aws = ((vy_C_point+vy_E_point)*(vx_C_point+vx_N_point) - (vy_S_point+vy_SE_point)*(vx_S_point+vx_C_point))/2*dely;
	aws += (gam/2*dely)*(d_absf(vy_C_point+vy_E_point)*(vx_C_point-vx_N_point) - d_absf(vy_S_point+vy_SE_point)*(vx_S_point-vx_C_point));
	return aws;
}

__device__ double del_sqr_vx_del_sqr_x(double delx, double vx_E_point, double vx_C_point, double vx_W_point){ //d²(vx)/dx²
	return (vx_E_point-2*vx_C_point+vx_W_point)/SQR(delx);
}

__device__ double del_sqr_vx_del_sqr_y(double dely, double vx_N_point, double vx_C_point, double vx_S_point){ //d²(vx)/dy²
	return (vx_N_point-2*vx_C_point+vx_S_point)/SQR(dely);
}

__device__ double del_vy_sqr_del_y(double gam, double dely, double vy_N_point, double vy_C_point, double vy_S_point){ // d(vy²)/dy
	double aws;
	aws = (SQR((vy_C_point+vy_N_point)/2) - SQR((vy_S_point+vy_C_point)/2))/dely;
	aws += (gam/dely)*(d_absf((vy_C_point+vy_N_point)/2)*((vy_C_point-vy_N_point)/2) - d_absf((vy_S_point+vy_C_point)/2)*((vy_S_point-vy_C_point)/2));
	return aws;
}

__device__ double del_vy_vx_del_x(double gam, double delx, double vy_W_point, double vy_C_point, double vy_E_point, double vx_N_point, double vx_C_point, double vx_W_point, double vx_NW_point){ //d(vx*vy)/dx
	double aws;
	aws = ((vx_C_point+vx_N_point)*(vy_C_point+vy_E_point) - (vx_W_point+vx_NW_point)*(vy_W_point+vy_C_point))/2*delx;
	aws += (gam/2*delx)*(d_absf(vx_C_point+vx_N_point)*(vy_C_point-vy_E_point) - d_absf(vx_W_point+vx_NW_point)*(vy_W_point-vy_C_point));
	return aws;
}

__device__ double del_sqr_vy_del_sqr_x(double delx, double vy_E_point, double vy_C_point, double vy_W_point){//d²(vy)/dx²
	return (vy_E_point-2*vy_C_point+vy_W_point)/SQR(delx);
}

__device__ double del_sqr_vy_del_sqr_y(double dely, double vy_N_point, double vy_C_point, double vy_S_point){//d²(vy)/dy²
	return (vy_N_point-2*vy_C_point+vy_S_point)/SQR(dely);
}

__global__ void comp_FG(int imax, int jmax, double gam, double delx, double dely, double Re, double gx, double gy, double del_time, double* d_vx, double* d_vy, double* d_F, double* d_G){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		double term1 = del_sqr_vx_del_sqr_x(delx, d_vx[idx+1], d_vx[idx], d_vx[idx-1]); 
								// del_sqr_vx_del_sqr_x(matrix[j][i+1].vx, matrix[j][i].vx,matrix[j][i-1].vx);
				
		double term2 = del_sqr_vx_del_sqr_y(dely, d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx+(imax+2)]); 
								// del_sqr_vx_del_sqr_y(matrix[j-1][i].vx, matrix[j][i].vx,matrix[j+1][i].vx);
				
		double term3 = del_vx_sqr_del_x(gam, delx, d_vx[idx], d_vx[idx-1], d_vx[idx+1]);
								// del_vx_sqr_del_x(matrix[j][i].vx,matrix[j][i-1].vx,matrix[j][i+1].vx);
				
		double term4 = del_vx_vy_del_y(gam, dely, d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx+(imax+2)], d_vy[idx], d_vy[idx+1], d_vy[idx+(imax+2)], d_vy[idx+(imax+2)+1]); 
								// del_vx_vy_del_y(matrix[j-1][i].vx,matrix[j][i].vx,matrix[j+1][i].vx, matrix[j][i].vy,matrix[j][i+1].vy,matrix[j+1][i].vy,matrix[j+1][i+1].vy);
		
		d_F[idx] = d_vx[idx] + del_time*(((term1+term2)/Re) - term3 - term4 + gx);
		//matrix[j][i].F=matrix[j][i].vx + del_time*(((term1+term2)/Re) - term3 - term4 + gx);	
	
	} else if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && (idx%(imax+2) == 0 || idx%(imax+2) == (imax+1)) ) {
		d_F[idx] = d_vx[idx]; //matrix[j][imax].F=matrix[j][imax].vx;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		
		double term5 = del_sqr_vy_del_sqr_x(delx, d_vy[idx+1], d_vy[idx], d_vy[idx-1]);		
								// del_sqr_vy_del_sqr_x(matrix[j][i+1].vy,matrix[j][i].vy,matrix[j][i-1].vy);
		
		double term6 = del_sqr_vy_del_sqr_y(dely, d_vy[idx-(imax+2)], d_vy[idx], d_vy[idx+(imax+2)]); 
								// del_sqr_vy_del_sqr_y(matrix[j-1][i].vy,matrix[j][i].vy,matrix[j+1][i].vy);
		
		double term7 = del_vy_vx_del_x(gam, delx, d_vy[idx-1], d_vy[idx], d_vy[idx+1], d_vx[idx-(imax+2)], d_vx[idx], d_vx[idx-1], d_vx[idx-(imax+2)-1]); 
								// del_vy_vx_del_x(matrix[j][i-1].vy, matrix[j][i].vy,matrix[j][i+1].vy, matrix[j-1][i].vx,matrix[j][i].vx, matrix[j][i-1].vx ,matrix[j-1][i-1].vx);
				
		double term8 = del_vy_sqr_del_y(gam, dely, d_vy[idx-(imax+2)], d_vy[idx], d_vy[idx+(imax+2)]);
								// del_vy_sqr_del_y(matrix[j-1][i].vy ,matrix[j][i].vy,matrix[j+1][i].vy);
		
		d_G[idx] = d_vy[idx] + del_time*(((term5+term6)/Re) - term7 - term8 + gy);
	
	} else if(idx < n && idx > (imax+1) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_G[idx] = d_vy[idx];
	}	
}

__global__ void comp_RHS(int imax, int jmax, double delx, double dely, double del_time, double* d_rhs, double* d_F, double* d_G){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;	
	int n = (imax+2)*(jmax+2);
	
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		d_rhs[idx] = ((d_F[idx]-d_F[idx-1])/delx +(d_G[idx]-d_G[idx+(imax+2)])/dely)/del_time;
	}	
}

__global__ void build_poisson_system(int jmax, int imax, double delx, double dely, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p, double* d_p_red, double* d_p_black){
	int i, j, idx;
	int ew, ee, es, en;
	for(j = jmax; j > 0; j--){
		es = j<jmax ? 1 : 0;
		en = j>1 ? 1 : 0;
		for(i = 1; i < imax+1; i++){
			idx = j*(imax+2)+i;
			ew = i>1 ? 1 : 0;
			ee = i<imax ? 1 : 0;
			d_diag_e[idx] = ee/SQR(delx);
			d_diag_w[idx] = ew/SQR(delx);
			d_diag_p[idx] = ((ee+ew)/SQR(delx) + (en+es)/SQR(dely));
			d_diag_n[idx] = en/SQR(dely);
			d_diag_s[idx] = es/SQR(dely);
		}
	}
}

__global__ void reductionMax(int imax, int jmax, double* d_partial, double* d_diff, double* d_flag){
	__shared__ 	double cache [REDCT_NUMTHREADS];
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double temp = 0;
	int n = (imax+2)*(jmax+2);
	int inc = blockDim.x*gridDim.x;
	int i, k;
	
	for(i = idx; i < n; i += inc){
		if(d_flag[i]){
			if(temp < d_diff[i]) temp = d_diff[i];
		}
	}
	
	cache[threadIdx.x] = temp;
	__syncthreads();
	
	for(k = (blockDim.x >> 1); k > 0; k >>= 1){
		if(threadIdx.x < k){
			if(cache[threadIdx.x] < cache[threadIdx.x+k]){
				cache[threadIdx.x] = cache[threadIdx.x+k];
			}
		}
		__syncthreads();
	}

	if(threadIdx.x == 0){
		d_partial[blockIdx.x] = cache[0];
	}
}

__global__ void red_SOR(int imax, int jmax, double omg, double* d_rhs, double* d_p_diff, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p, double* d_flag, double* d_p_red, double* d_p_black){	
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // idx = (imax+2)
	int size = (imax+2)*(jmax+2);
	int new_imax = (imax+2)/2;
	int line = idx / new_imax; // line = 1
	int paridade = line % 2; // paridade = 0
	double aux;
	
	int nidx_red = 2*idx + (idx/new_imax)%2;
	int nidx_black = 2*idx + ((idx/new_imax + 1))%2;

	// __shared__ double d_buffer[TAM_BUFFER_SOR*4];

	// if(threadIdx.x == TAM_BUFFER_SOR*3 - 1) {
	// 	d_buffer[threadIdx.x + TAM_BUFFER_SOR] = d_p_black[idx + new_imax];
	// 	// if(threadIdx.x + TAM_BUFFER_SOR > TAM_BUFFER_SOR*4-1) printf("max = | id =\n");
	// 	if(idx + new_imax > size/2) printf("max = | id =\n");
	// }
	// d_buffer[threadIdx.x] = d_p_black[idx];

	// __syncthreads();

	// if(idx == 0) printf("blockDim.x = %d\n", blockDim.x);
	// idx = j*(imax+2)+i;
	// idx+1 = j*(imax+2)+i+1;
	// idx+1+(imax+2) = j*(imax+2)+(imax+2)+(i+1);
	// idx+1+(imax+2) = (j+1)*(imax+2) +(i+1);

	if(	idx < size/2 && nidx_red < size && nidx_red > (imax+1) && nidx_red < (imax+2)*(jmax+1) && nidx_red % (imax+2) > 0 && nidx_red % (imax+2) < (imax+1)){

		aux = d_diag_s[nidx_red]*d_p_black[idx + new_imax] + d_diag_n[nidx_red]*d_p_black[idx - new_imax]+ d_diag_e[nidx_red]*d_p_black[idx+paridade] + d_diag_w[nidx_red]*d_p_black[idx-1+paridade];
		aux = (1-omg)*d_p_red[idx] + omg*(aux-d_rhs[nidx_red])/d_diag_p[nidx_red];
		d_p_diff[nidx_red] = d_absf(aux-d_p_red[idx]);		
		d_flag[nidx_red] = 1;
		d_p_red[idx] = aux;
	}

	if(	nidx_black < size) d_flag[nidx_black] = 0;
}

__global__ void black_SOR(int imax, int jmax, double omg, double* d_rhs, double* d_p_diff, double* d_diag_n, double* d_diag_s, double* d_diag_e, double* d_diag_w, double* d_diag_p, double* d_flag, double* d_p_red, double* d_p_black){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int size = (imax+2)*(jmax+2);
	int new_imax = (imax+2)/2;
	int line = idx / new_imax;
	int paridade =!(line % 2);
	double aux;
	
	int nidx_black = 2*idx + ((idx/new_imax + 1))%2;
	int nidx_red = 2*idx + (idx/new_imax)%2;

	if(idx < size/2 && nidx_black < size && nidx_black > (imax+1) && nidx_black < (imax+2)*(jmax+1) && nidx_black % (imax+2) > 0 && nidx_black % (imax+2) < (imax+1)){

		aux = d_diag_s[nidx_black]*d_p_red[idx + new_imax] + d_diag_n[nidx_black]*d_p_red[idx - new_imax]+ d_diag_e[nidx_black]*d_p_red[idx+paridade] + d_diag_w[nidx_black]*d_p_red[idx-1+paridade];
		aux = (1-omg)*d_p_black[idx] + omg*(aux-d_rhs[nidx_black])/d_diag_p[nidx_black];
		d_p_diff[nidx_black] = d_absf(aux-d_p_black[idx]);		
		d_flag[nidx_black] = 1;
		d_p_black[idx] = aux;
	}
	
	if(	nidx_red < size) d_flag[nidx_red] = 0;
}

__global__ void check_diff(double* d_partial, double* d_diff, double* diff) {
	__shared__ double diff_local[1];
	diff_local[0] = 0.0;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(d_partial[idx] > diff_local[0]) diff_local[0] = d_partial[idx];

	// printf("diff = %lf\n", diff[0]);
	__syncthreads();
	if(idx == 0) {
		*d_diff = diff_local[0];
		// printf("diff_device = %lf\n", diff_local[0]);
	}
}

int calc_interval(void* start, void* end) {
	struct timeb *start_time = (timeb*)start;
	struct timeb *end_time = (timeb*)end;
	ftime(end_time);

	int total_time = (int) (1000.0 * (end_time->time - start_time->time)
			+ (end_time->millitm - start_time->millitm));

	return total_time;
}

int Poisson(){
	int iter = 0;
	int i;
	double partial[NUMBLOCKS];
	double diff = 0;
	struct timeb start_rbsor, end_rbsor, start_redct, end_redct, start_aux, end_aux, start_total, end_total;
	int total_rbsor = 0, total_redct = 0, total_aux = 0, total = 0;

	// int n_blocos_sor = ((imax+2)*(jmax+2)/2)/TAM_BUFFER_SOR*3;
	// printf("%d == %d\n", n_blocos_sor * TAM_BUFFER_SOR, (imax)*(jmax));
	
	ftime(&start_total);
	while(iter < max_iter){
		ftime(&start_rbsor);
		for(i = 0; i < n_iter; i++){
			red_SOR<<<n_blocos, n_threads>>>(imax, jmax, omg, d_rhs, d_p_diff, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p, d_flag, d_p_red, d_p_black);
			black_SOR<<<n_blocos, n_threads>>>(imax, jmax, omg, d_rhs, d_p_diff, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p, d_flag, d_p_red, d_p_black);
		}

		ftime(&start_redct);
		diff = 0;
		reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_p_diff, d_flag);
		total_rbsor += calc_interval(&start_rbsor, &end_rbsor);
		
		err = hipMemcpy(partial, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// if(iter > 500) {
		// 	iter += n_iter;
		// 	continue;
		// }

		total_redct += calc_interval(&start_redct, &end_redct);
		ftime(&start_aux);

		for(i = 0; i < NUMBLOCKS; i++){
			if(partial[i] > diff) diff = partial[i];
		}
		total_aux += calc_interval(&start_aux, &end_aux);
		iter += n_iter;
		
		if(diff < eps){
			break;
			// return iter;
		}
	}
	// printf("diff = %lf\n", diff);

	ftime(&end_total);
	total += (int) (1000.0 * (end_total.time - start_total.time)
				+ (end_total.millitm - start_total.millitm));

	// printf("Total time - reductionMax: %d milliseconds\n", total_redct);
	// printf("Total time - redBlackSOR: %d milliseconds\n", total_rbsor);
	// printf("Total time - aux: %d milliseconds\n", total_aux);
	// printf("Total time - total: %d milliseconds\n", total);
	return iter;
}

__global__ void d_adap_Vel(int imax, int jmax, double delx, double dely, double del_time, double* d_vx, double* d_vy, double* d_F, double* d_G, double* d_vxdiff, double* d_vydiff, double* d_vxflag, double* d_vyflag, double* d_p_red, double* d_p_black){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int n = (imax+2)*(jmax+2);
	int line = idx / (imax+2);
	int paridade = line % 2;

	double dp_nidx = idx % 2 == paridade ? d_p_red[idx/2] : d_p_black[idx/2];
	double dp_nidx_i = (idx+1) % 2 == paridade ? d_p_red[(idx+1)/2] : d_p_black[(idx+1)/2];
	double dp_nidx_imax = idx % 2 == paridade ? d_p_black[(idx-(imax+2))/2] : d_p_red[(idx-(imax+2))/2];
	
	double aux;
	if(idx < n && idx > (imax+1) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax)){
		aux = d_F[idx] - (del_time*(dp_nidx_i - dp_nidx)/delx); //matrix[j][i].F - (del_time*(matrix[j][i+1].p - matrix[j][i].p)/delx);
		d_vxdiff[idx] = d_absf(aux-d_vx[idx]);   //absf(aux-matrix[j][i].vx);
		d_vxflag[idx] = 1;
		d_vx[idx] = aux;		
	} else{
		d_vxflag[idx] = 0;
	}
	
	if(idx < n && idx >= 2*(imax+2) && idx < (jmax+1)*(imax+2) && idx%(imax+2) > 0 && idx%(imax+2) < (imax+1)){
		aux = d_G[idx] - (del_time*(dp_nidx_imax - dp_nidx)/dely); //matrix[j][i].G - (del_time*(matrix[j-1][i].p - matrix[j][i].p)/dely);
		d_vydiff[idx] = d_absf(aux-d_vy[idx]);//absf(aux-matrix[j][i].vy);
		d_vyflag[idx] = 1;
		d_vy[idx] = aux;
	}	else{
		d_vyflag[idx] = 0;
	}	
}

int adap_Vel(int n_blocos, int n_threads){
	double diffvx = 0;
	double diffvy = 0;
	int i;
	
	double partialvx[NUMBLOCKS];
	double partialvy[NUMBLOCKS];
	
	d_adap_Vel<<< n_blocos, n_threads >>>(imax, jmax, delx, dely, del_time, d_vx, d_vy, d_F, d_G, d_vxdiff, d_vydiff, d_vxflag, d_vyflag, d_p_red, d_p_black);		

	reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_vxdiff, d_vxflag);
	err = hipMemcpy(partialvx, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	reductionMax<<<REDCT_NUMTHREADS, NUMBLOCKS>>>(imax, jmax, d_partial, d_vydiff, d_vyflag);
	err = hipMemcpy(partialvy, d_partial, NUMBLOCKS*sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess)	{
		fprintf(stderr, "Failed to copy pointer d_partial from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	for(i = 0; i < NUMBLOCKS; i++){
		if(partialvy[i] > diffvy) diffvy = partialvy[i];
		if(partialvx[i] > diffvx) diffvx = partialvx[i];
	}
	
	if(diffvy < eps && diffvx < eps) return 1;
	return 0;
}

int main(int argc, char ** argv){
	read_file(argv[1]);
	alocate_vectors_host();
	alocate_vectors_device();
	copy_vectors_host_to_device();
    
	n_threads = (imax+2);
	n_blocos = ((imax+2)*(jmax+2)+ n_threads-1)/n_threads;
	init_UVP<<<n_blocos, n_threads>>>(imax, jmax, vx_init, vy_init, p_init, d_vx, d_vy, d_p, d_rhs, d_F, d_G, d_p_red, d_p_black);
	build_poisson_system<<<1, 1>>>(jmax, imax, delx, dely, d_diag_n, d_diag_s, d_diag_e, d_diag_w, d_diag_p, d_p_red, d_p_black);
	
	hipArray* cuArray;

	hipMallocArray(&cuArray, &tex.channelDesc, n_threads*n_threads/2, 1);
	hipBindTextureToArray(tex, cuArray);

	tex.filterMode = hipFilterModeLinear;

	int set_time = 1;
	double ant_del_time = 1.0;
	double eps_time = 1e-7;
	int num_time = 0;
	int limit = 100;
	int max_frames = 2000;
	int dyn_eps_flag = 0;

  // gettimeofday(&start, NULL);
	ftime(&start);

	int frame = 0, frames = 0;
	float last_time_frame = 0.0, last_frame_time = 0.0;
	printf("Entrando no laco principal\n");
	while(!state){
		ftime(&aux_start);
		if(set_time){
			comp_delt();
		}
		if(absf(ant_del_time-del_time) < eps_time){
			num_time++;
			if(num_time == limit){
				set_time = 0;
			}
		}	else{
			num_time = 0;
		}
		
		set_bondCond();
		set_lidDrivenCavityProblem<<<n_blocos, n_threads>>>(1.0, imax, jmax, d_vx);
		
		comp_FG<<<n_blocos, n_threads>>>(imax, jmax, gam, delx, dely, Re, gx, gy, del_time, d_vx, d_vy, d_F, d_G);
		comp_RHS<<<n_blocos, n_threads>>>(imax, jmax, delx, dely, del_time, d_rhs, d_F, d_G);		

		Poisson();

		// if(time_frame < 1000) break;
		// printf("Time = %lf/%lf\r", ttime, final_time);
		
		state = adap_Vel(n_blocos, n_threads);
		ftime(&end);
		
		int time_frame = (int) (1000.0 * (end.time - aux_start.time)
        + (end.millitm - aux_start.millitm));
		ttime += del_time;
		ant_del_time = del_time;

		frame++; frames++;
		ftime(&end);
		time_frame = (int) (1000.0 * (end.time - start.time)
        + (end.millitm - start.millitm));

		// printf("Total time elapsed: %lf seconds - Iteracoes: %d\n", time_frame/1000.0, frames);
		// printf("Time = %d\n", time_frame);
		// if(time_frame - last_frame_time > 1000.0/24.0 && eps <= 0.001 && !dyn_eps_flag){
		// 	double range = absf(1000.0/24.0 - (time_frame - last_frame_time));
		// 	// printf("range = %lf\n", log2(range));
		// 	eps *= 2.0;//log(range);
		// 	eps = min(0.001, eps);
		// }
		// if(time_frame - last_frame_time < 1000.0/30.0 && !dyn_eps_flag){
		// 	eps /= 1.1;
		// 	if(eps <= 1e-7) dyn_eps_flag = 1;
		// }
		if(time_frame - last_time_frame >= 1000.0) {
			printf("============ %d fps\r", frame);
			frame = 0;
			last_time_frame = time_frame;
		}

		last_frame_time = time_frame;
		// break;
	}
	
	set_bondCond();		
	set_lidDrivenCavityProblem<<<n_blocos, n_threads>>>(1.0, imax, jmax, d_vx);
	// gettimeofday(&end, NULL);
	ftime(&end);

	int milliseconds = (int) (1000.0 * (end.time - start.time) + (end.millitm - start.millitm));
	
	// printf("Time = %lf/%lf\n", ttime, final_time);
	printf("Total time elapsed: %lf seconds\n\n", milliseconds/1000.0);

	copy_vectors_device_to_host(); 
  write_file(argv[2]);
    
  free_vectors_device();	
	free_vectors_host();

	return 0;
}
